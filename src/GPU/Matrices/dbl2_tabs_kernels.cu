#include "hip/hip_runtime.h"
/* The file dbl2_tabs_kernels.cu defines the functions specified in
 * the file dbl2_tabs_kernels.h. */

#include <iostream>
#ifdef gpufun
#include "double_double_gpufun.cu"
#endif
#include "dbl2_tabs_kernels.h"

using namespace std;

__global__ void dbl2_small_invert_upper 
( int dim, double *Uhi, double *Ulo, double *invUhi, double *invUlo )
{
   const int k = threadIdx.x; // thread k computes k-th column of inverse

   __shared__ double Ucolhi[dd_shmemsize];
   __shared__ double Ucollo[dd_shmemsize];
   __shared__ double invUrowshi[dd_shmemsize];
   __shared__ double invUrowslo[dd_shmemsize];

   double rhshi,rhslo,xvalhi,xvallo,acchi,acclo;

   int colidx = dim*(dim-1);          // start with the last column

   Ucolhi[k] = Uhi[colidx+k];         // load the last column
   Ucollo[k] = Ulo[colidx+k];
   rhshi = ((double) int(k == dim-1));  // right hand side for each thread
   rhslo = 0.0;
   int rowidx = (dim - 1)*dim + k;      // the row index in the inverse

   __syncthreads();
   // invUrows[rowidx] = rhs/Ucol[k]; // last row of the inverse
   ddg_div(rhshi,rhslo,Ucolhi[k],Ucollo[k],
           &invUrowshi[rowidx],&invUrowslo[rowidx]);

   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhshi = ((double) int(k == i)); // set rhs for i-th unit vector
      rhslo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = dim*j;              // need column j of U

         Ucolhi[k] = Uhi[colidx+k];
         Ucollo[k] = Ulo[colidx+k];

         rowidx = j*dim + k;          // need solution value

         xvalhi = invUrowshi[rowidx];
         xvallo = invUrowslo[rowidx];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval; // update right hand side
         ddg_mul(Ucolhi[i],Ucollo[i],xvalhi,xvallo,&acchi,&acclo);
         ddg_dec(&rhshi,&rhslo,acchi,acclo);
      }
      rowidx = i*dim + k;             // save in i-th row of inverse

      colidx = dim*i;                 // need column i of U
      Ucolhi[k] = Uhi[colidx+k];
      Ucollo[k] = Ulo[colidx+k];

      __syncthreads();
      // invUrows[rowidx] = rhs/Ucol[i];
      ddg_div(rhshi,rhslo,Ucolhi[i],Ucollo[i],
              &invUrowshi[rowidx],&invUrowslo[rowidx]);
   }
   rowidx = 0;
   for(int i=0; i<dim; i++)
   {
      __syncthreads();
      invUhi[rowidx+k] = invUrowshi[rowidx+k];
      invUlo[rowidx+k] = invUrowslo[rowidx+k];
      rowidx = rowidx + dim;
   }
}

__global__ void dbl2_medium_invert_upper
 ( int dim, double *Uhi, double *Ulo, double *invUhi, double *invUlo)
{
   const int k = threadIdx.x;  // thread k computes k-th column of inverse

   __shared__ double Ucolhi[dd_shmemsize];      // one column of U
   __shared__ double Ucollo[dd_shmemsize];      // one column of U
   __shared__ double invUrowhi[dd_shmemsize];   // one row of invU
   __shared__ double invUrowlo[dd_shmemsize];   // one row of invU

   double rhshi,rhslo,xvalhi,xvallo,acchi,acclo;

   int colidx = dim*(dim-1);           // start with the last column

   Ucolhi[k] = Uhi[colidx+k];          // load the last column
   Ucollo[k] = Ulo[colidx+k];
   rhshi = ((double) int(k == dim-1)); // right hand side for each thread
   rhslo = 0.0;
   int rowidx = (dim - 1)*dim + k;     // the row index in the inverse

   // invUrow[k] = rhs/Ucol[k];          // last row of the inverse
   ddg_div(rhshi,rhslo,Ucolhi[k],Ucollo[k],&invUrowhi[k],&invUrowlo[k]);
   invUhi[rowidx] = invUrowhi[k];     // store the last row into invU
   invUlo[rowidx] = invUrowlo[k]; 

   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhshi = ((double) int(k == i)); // set rhs for i-th unit vector
      rhslo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = dim*j;              // need column j of U
         Ucolhi[k] = Uhi[colidx+k];
         Ucollo[k] = Ulo[colidx+k];

         rowidx = j*dim + k;            // need solution value
         invUrowhi[k] = invUhi[rowidx]; // load invU row into invUrow
         invUrowlo[k] = invUlo[rowidx];
         xvalhi = invUrowhi[k];
         xvallo = invUrowlo[k];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval;    // update right hand side
         ddg_mul(Ucolhi[i],Ucollo[i],xvalhi,xvallo,&acchi,&acclo);
         ddg_dec(&rhshi,&rhslo,acchi,acclo);
      }
      colidx = dim*i;                 // need column i of U
      Ucolhi[k] = Uhi[colidx+k];
      Ucollo[k] = Ulo[colidx+k];
      rowidx = i*dim + k;             // save in i-th row of inverse

      __syncthreads();
      // invUrow[k] = rhs/Ucol[i];
      ddg_div(rhshi,rhslo,Ucolhi[i],Ucollo[i],&invUrowhi[k],&invUrowlo[k]);
      invUhi[rowidx] = invUrowhi[k];
      invUlo[rowidx] = invUrowlo[k];
   }
}

__global__ void  dbl2_invert_tiles
 ( int dim, double *Uhi, double *Ulo, double *invUhi, double *invUlo )
{
   const int B = blockIdx.x;   // block index
   const int k = threadIdx.x;  // thread k computes k-th column of inverse
   const int offset = dim*dim*B; // offset in U and invU

   __shared__ double Ucolhi[dd_shmemsize];      // one column of U
   __shared__ double Ucollo[dd_shmemsize];
   __shared__ double invUrowhi[dd_shmemsize];   // one row of invU
   __shared__ double invUrowlo[dd_shmemsize]; 

   double rhshi,rhslo,xvalhi,xvallo,acchi,acclo;

   int colidx = offset + dim*(dim-1); // start with the last column

   Ucolhi[k] = Uhi[colidx+k];         // load the last column
   Ucollo[k] = Ulo[colidx+k];
   rhshi = ((double) int(k == dim-1));  // right hand side for each thread
   rhslo = 0.0;
   int rowidx = offset + (dim - 1)*dim + k; // row index in the inverse

   // invUrow[k] = rhs/Ucol[k];       // last row of the inverse
   ddg_div(rhshi,rhslo,Ucolhi[k],Ucollo[k],&invUrowhi[k],&invUrowlo[k]);
   invUhi[rowidx] = invUrowhi[k];     // store the last row into invU
   invUlo[rowidx] = invUrowlo[k];

   for(int i=dim-2; i>=0; i--)        // compute row with index i
   {
      rhshi = ((double) int(k == i));   // set rhs for i-th unit vector
      rhslo = 0.0;

      for(int j=i+1; j<dim; j++)
      {
         colidx = offset + dim*j;     // need column j of U
         Ucolhi[k] = Uhi[colidx+k];
         Ucollo[k] = Ulo[colidx+k];

         rowidx = offset + j*dim + k; // need solution value
         invUrowhi[k] = invUhi[rowidx]; // load invU row into invUrow
         invUrowlo[k] = invUlo[rowidx]; // load invU row into invUrow
         xvalhi = invUrowhi[k];
         xvallo = invUrowlo[k];

         __syncthreads();
         // rhs = rhs - Ucol[i]*xval;    // update right hand side
         ddg_mul(Ucolhi[i],Ucollo[i],xvalhi,xvallo,&acchi,&acclo);
         ddg_dec(&rhshi,&rhslo,acchi,acclo);
      }
      colidx = offset + dim*i;        // need column i of U
      Ucolhi[k] = Uhi[colidx+k];
      Ucollo[k] = Ulo[colidx+k];
      rowidx = offset + i*dim + k;    // save in i-th row of inverse

      __syncthreads();
      // invUrow[k] = rhs/Ucol[i];
      ddg_div(rhshi,rhslo,Ucolhi[i],Ucollo[i],&invUrowhi[k],&invUrowlo[k]);
      invUhi[rowidx] = invUrowhi[k];
      invUlo[rowidx] = invUrowlo[k];
   }
}

__global__ void dbl2_multiply_inverse
 ( int dim, int idx, double *invUhi, double *invUlo,
   double *whi, double *wlo )
{
   const int k = threadIdx.x;     // thread k computes k-th product
   const int rhsoff = dim*idx;    // offset for the right hand size
   const int offset = dim*rhsoff; // offset for diagonal tile

   __shared__ double workhi[dd_shmemsize];      // copy of w
   __shared__ double worklo[dd_shmemsize];      // copy of w

   workhi[k] = whi[rhsoff+k];
   worklo[k] = wlo[rhsoff+k];

   double resulthi = 0.0; // each thread stores its product in result
   double resultlo = 0.0;
   double coeffhi,coefflo,acchi,acclo;

   for(int j=0; j<dim; j++)  // column j of the inverse diagonal tile
   {
      coeffhi = invUhi[offset+k*dim+j]; // thread k does row k
      coefflo = invUlo[offset+k*dim+j];
      // result = result + coeff*work[j];
      ddg_mul(coeffhi,coefflo,workhi[j],worklo[j],&acchi,&acclo);
      ddg_inc(&resulthi,&resultlo,acchi,acclo);
   }
   whi[rhsoff+k] = resulthi;
   wlo[rhsoff+k] = resultlo;
}

__global__ void dbl2_back_substitute
 ( int dim, int idx, double *Uhi, double *Ulo, double *whi, double *wlo )
{
   const int B = blockIdx.x;     // block index
   const int k = threadIdx.x;    // thread k computes k-th product
   const int offset = B*dim*dim; // numbers to skip

   __shared__ double wrkhi[dd_shmemsize];   // copy of w
   __shared__ double wrklo[dd_shmemsize]; 
   __shared__ double solhi[dd_shmemsize];    // solution to update with
   __shared__ double sollo[dd_shmemsize];

   wrkhi[k] = whi[B*dim+k];    // block B updates B-th slice of w
   wrklo[k] = wlo[B*dim+k];
   solhi[k] = whi[idx*dim+k];  // solution that is back substituted
   sollo[k] = wlo[idx*dim+k];

   double resulthi = 0.0; // each thread stores its product in result
   double resultlo = 0.0;
   double coeffhi,coefflo,acchi,acclo;

   for(int j=0; j<dim; j++)  // column j of the inverse diagonal tile
   {
      coeffhi = Uhi[offset+k*dim+j];
      coefflo = Ulo[offset+k*dim+j];
      // result = result + coeff*sol[j];
      ddg_mul(coeffhi,coefflo,solhi[j],sollo[j],&acchi,&acclo);
      ddg_inc(&resulthi,&resultlo,acchi,acclo);
   }
   // wrk[k] = wrk[k] - result; // subtract product
   ddg_dec(&wrkhi[k],&wrklo[k],resulthi,resultlo);
   whi[B*dim+k] = wrkhi[k];
   wlo[B*dim+k] = wrklo[k];
}

void GPU_dbl2_upper_inverse
 ( int dim, double **Uhi, double **Ulo, double **invUhi, double **invUlo )
{
   const int szU = dim*dim;

   double *Uhi_h = new double[szU];     // Uhi_h stores the columns of Uhi
   double *Ulo_h = new double[szU];     // Ulo_h stores the columns of Ulo 
   double *Uhi_d;                       // Uhi_d is Uhi_h on the device
   double *Ulo_d;                       // Ulo_d is Ulo_h on the device
   double *invUhi_h = new double[szU];  // high doubles of the inverse
   double *invUlo_h = new double[szU];  // low doubles of the inverse
   double *invUhi_d;                    // invUhi_d is invUhi_h on the device
   double *invUlo_d;                    // invUlo_d is invUlo_h on the device

   int ix = 0;
   for(int j=0; j<dim; j++)
      for(int i=0; i<dim; i++)
      {
         Uhi_h[ix]   = Uhi[i][j];
         Ulo_h[ix++] = Ulo[i][j];
      }

   // only for debugging
   // test_dbl2_small_invert_upper(dim,Uhi_h,Ulo_h,invUhi,invUlo_h);

   size_t szmat = szU*sizeof(double);
   hipMalloc((void**)&Uhi_d,szmat);
   hipMalloc((void**)&Ulo_d,szmat);
   hipMalloc((void**)&invUhi_d,szmat);
   hipMalloc((void**)&invUlo_d,szmat);
   hipMemcpy(Uhi_d,Uhi_h,szmat,hipMemcpyHostToDevice);
   hipMemcpy(Ulo_d,Ulo_h,szmat,hipMemcpyHostToDevice);

   if(dim <= 16)
      dbl2_small_invert_upper<<<1,dim>>>(dim,Uhi_d,Ulo_d,invUhi_d,invUlo_d);
   else
      dbl2_medium_invert_upper<<<1,dim>>>(dim,Uhi_d,Ulo_d,invUhi_d,invUlo_d);

   hipMemcpy(invUhi_h,invUhi_d,szmat,hipMemcpyDeviceToHost);
   hipMemcpy(invUlo_h,invUlo_d,szmat,hipMemcpyDeviceToHost);

   ix = 0;
   for(int i=0; i<dim; i++)
      for(int j=0; j<dim; j++)
      {
         invUhi[i][j] = invUhi_h[ix];
         invUlo[i][j] = invUlo_h[ix++];
      }

   free(Uhi_h); free(invUhi_h);
   free(Ulo_h); free(invUlo_h);
}

void GPU_dbl2_upper_tiled_solver
 ( int dim, int szt, int nbt, double **Uhi, double **Ulo,
   double *bhi, double *blo, double *xhi, double *xlo )
{
   const int nbr = nbt*szt*szt;   // number of doubles on diagonal tiles
   double *Dhi_h = new double[nbr];    // the diagonal tiles on the host
   double *Dlo_h = new double[nbr];    // low doubles of diagonal tiles
   double *Dhi_d;                      // diagonal tiles on the device
   double *Dlo_d;                      // low doubles of diagonal tiles
   double *invDhi_h = new double[nbr]; // inverse of diagonal tiles on host 
   double *invDlo_h = new double[nbr]; // low doubles of inverse tiles
   double *invDhi_d;                   // invDhi_d is invDhi_h on device
   double *invDlo_d;                   // invDlo_d is invDlo_h on device
   int offset;
   int ix = 0;

   for(int k=0; k<nbt; k++) // copy columns of the k-th tile
   {
      offset = k*szt;
      for(int j=0; j<szt; j++)
         for(int i=0; i<szt; i++)
         {
            Dhi_h[ix]   = Uhi[offset+i][offset+j];
            Dlo_h[ix++] = Ulo[offset+i][offset+j];
         }
   }
   const size_t sznum = nbr*sizeof(double);
   hipMalloc((void**)&Dhi_d,sznum);
   hipMalloc((void**)&Dlo_d,sznum);
   hipMalloc((void**)&invDhi_d,sznum);
   hipMalloc((void**)&invDlo_d,sznum);
   hipMemcpy(Dhi_d,Dhi_h,sznum,hipMemcpyHostToDevice);
   hipMemcpy(Dlo_d,Dlo_h,sznum,hipMemcpyHostToDevice);

   dbl2_invert_tiles<<<nbt,szt>>>(szt,Dhi_d,Dlo_d,invDhi_d,invDlo_d);

   double *rhshi_d;                    // right hand side on device
   double *rhslo_d;
   const size_t szrhs = dim*sizeof(double);
   hipMalloc((void**)&rhshi_d,szrhs);
   hipMalloc((void**)&rhslo_d,szrhs);
   hipMemcpy(rhshi_d,bhi,szrhs,hipMemcpyHostToDevice);
   hipMemcpy(rhslo_d,blo,szrhs,hipMemcpyHostToDevice);

   dbl2_multiply_inverse<<<1,szt>>>
      (szt,nbt-1,invDhi_d,invDlo_d,rhshi_d,rhslo_d);

   int nbrUcol = (nbt-1)*szt*szt;           // #doubles in column of U
   double *Ucolhi_h = new double[nbrUcol];  // column of U on host
   double *Ucollo_h = new double[nbrUcol];  // column of U on host
   double *Ucolhi_d;
   double *Ucollo_d;
   const size_t szUcol = nbrUcol*sizeof(double);
   hipMalloc((void**)&Ucolhi_d,szUcol);
   hipMalloc((void**)&Ucollo_d,szUcol);

   int coloff,rowoff;

   for(int k=nbt-1; k>0; k--)      // update with solution tile k
   {
      coloff = k*szt;      // column offset to update with solution tile k
      ix = 0;
      for(int L=0; L<k; L++)       // copy k tiles of U
      {
         rowoff = L*szt;           // row offset for update data
         for(int i=0; i<szt; i++)
            for(int j=0; j<szt; j++)
            {
               Ucolhi_h[ix]   = Uhi[rowoff+i][coloff+j];
               Ucollo_h[ix++] = Ulo[rowoff+i][coloff+j];
            }
      }
      hipMemcpy(Ucolhi_d,Ucolhi_h,nbrUcol*sizeof(double),
                 hipMemcpyHostToDevice);
      hipMemcpy(Ucollo_d,Ucollo_h,nbrUcol*sizeof(double),
                 hipMemcpyHostToDevice);

      dbl2_back_substitute<<<k,szt>>>
         (szt,k,Ucolhi_d,Ucollo_d,rhshi_d,rhslo_d);

      // (k-1)-th solution tile is ready for inverse multiplication
      dbl2_multiply_inverse<<<1,szt>>>
         (szt,k-1,invDhi_d,invDlo_d,rhshi_d,rhslo_d);

      nbrUcol = nbrUcol - szt*szt; // one tile less used in update
   }
   hipMemcpy(xhi,rhshi_d,szrhs,hipMemcpyDeviceToHost);
   hipMemcpy(xlo,rhslo_d,szrhs,hipMemcpyDeviceToHost);

   // copy of invD_d is needed only for testing purposes
   hipMemcpy(invDhi_h,invDhi_d,sznum,hipMemcpyDeviceToHost);
   hipMemcpy(invDlo_h,invDlo_d,sznum,hipMemcpyDeviceToHost);

   ix = 0;
   for(int k=0; k<nbt; k++) // copy rows of the inverse of the k-th tile
   {
      offset = k*szt;
      for(int i=0; i<szt; i++)
         for(int j=0; j<szt; j++)
         {
            Uhi[offset+i][offset+j] = invDhi_h[ix];
            Ulo[offset+i][offset+j] = invDlo_h[ix++];
         }
   }
   free(Dhi_h); free(invDhi_h); free(Ucolhi_h);
   free(Dlo_h); free(invDlo_h); free(Ucollo_h);
}
